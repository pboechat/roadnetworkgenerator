#include "hip/hip_runtime.h"
#include <RoadNetworkGraphGenerator.h>
#include <Constants.h>
#include <CpuGpuCompatibility.h>
#include <Procedures.h>
#include <Road.h>
#include <Branch.h>
#include <Quadtree.h>
#include <Quadrant.h>
#include <QuadrantEdges.h>
#include <Primitive.h>
#include <BaseGraph.h>
#include <Graph.h>
#include <Box2D.h>
#include <MinimalCycleBasis.h>
#include <ConvexHull.h>
#include <OBB2D.h>
#include <VectorMath.h>
#include <Context.cuh>
#include <WorkQueue.cuh>
#include <PseudoRandomNumbers.cuh>
#include <ExpansionKernel.cuh>
#include <CollisionDetectionKernel.cuh>
#include <Timer.h>
#include <GlobalVariables.cuh>
#include <GlobalVariables.h>
#include <Log.h>

#include <hiprand.h>

#include <exception>
#include <memory>

#define SAFE_MALLOC_ON_HOST(__variable, __type, __amount) \
	__variable = 0; \
	__variable = (__type*)malloc(sizeof(__type) * __amount); \
	if (__variable == 0) \
	{ \
		throw std::exception(#__variable": insufficient memory"); \
	}

#define CREATE_CPU_TIMER(x) \
	float elapsedTime_##x = 0.0f; \
	Timer timer_##x
#define START_CPU_TIMER(x) timer_##x.start()
#define STOP_CPU_TIMER(x) \
	timer_##x##.end(); \
	elapsedTime_##x += timer_##x##.elapsedTime()
#define GET_CPU_TIMER_ELAPSED_TIME(x, y) y = timer_##x##.elapsedTime()
#define DESTROY_CPU_TIMER(x)

#ifdef USE_CUDA
#include <cutil.h>
#include <cutil_timer.h>
#define SAFE_MALLOC_ON_DEVICE(__variable, __type, __amount) cudaCheckedCall(hipMalloc((void**)&__variable, sizeof(__type) * __amount))
#define SAFE_MALLOC_PITCH_ON_DEVICE(__variable, __type, __pitch, __width, __height) cudaCheckedCall(hipMallocPitch((void**)&__variable, &__pitch, sizeof(__type) * __width, sizeof(__type) * __height))
#define SAFE_FREE_ON_DEVICE(__variable) cudaCheckedCall(hipFree(__variable))
#define MEMCPY_TO_SYMBOL(__destination, __source, __size) cudaCheckedCall(hipMemcpyToSymbol(HIP_SYMBOL(__destination), __source, __size))
#define MEMCPY_HOST_TO_DEVICE(__destination, __source, __size) cudaCheckedCall(hipMemcpy(__destination, __source, __size, hipMemcpyHostToDevice))
#define MEMCPY2D_HOST_TO_DEVICE(__destination, __source, __hostPitch, __devicePitch, __width, __height) cudaCheckedCall(hipMemcpy2D(__destination, __devicePitch, __source, __hostPitch, __width, __height, hipMemcpyHostToDevice))
#define MEMCPY_DEVICE_TO_DEVICE(__destination, __source, __size) cudaCheckedCall(hipMemcpy(__destination, __source, __size, hipMemcpyDeviceToDevice))
#define MEMCPY_DEVICE_TO_HOST(__destination, __source, __size) cudaCheckedCall(hipMemcpy(__destination, __source, __size, hipMemcpyDeviceToHost))
#define MEMSET_ON_DEVICE(__variable, __value, __size) cudaCheckedCall(hipMemset(__variable, __value, __size))
#define BIND_AS_TEXTURE2D(__deviceVariable, __texture, __pitch, __width, __height) \
	{ \
		__texture.filterMode = hipFilterModePoint; \
		__texture.addressMode[0]    = hipAddressModeClamp; \
		__texture.addressMode[1]    = hipAddressModeClamp; \
		__texture.normalized = false; \
		hipChannelFormatDesc descriptor = hipCreateChannelDesc(sizeof(unsigned char) << 3, 0, 0, 0, hipChannelFormatKindUnsigned); \
		cudaCheckedCall(hipBindTexture2D(0, __texture, __deviceVariable, descriptor, __width, __height, __pitch)); \
	}
#define UNBIND_TEXTURE2D(__texture) cudaCheckedCall(hipUnbindTexture(__texture))
#define INVOKE_GLOBAL_CODE(__function, __numBlocks, __numThreads) \
	__function<<<__numBlocks, __numThreads>>>(); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE1(__function, __numBlocks, __numThreads, __arg1) \
	__function<<<__numBlocks, __numThreads>>>(__arg1); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE2(__function, __numBlocks, __numThreads, __arg1, __arg2) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE3(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE4(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE5(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4, __arg5); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE6(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE7(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE8(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE9(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE10(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE11(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10, __arg11) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10, __arg11); \
	cudaCheckError()
#define INVOKE_GLOBAL_CODE12(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10, __arg11, __arg12) \
	__function<<<__numBlocks, __numThreads>>>(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10, __arg11, __arg12); \
	cudaCheckError()
#define CREATE_GPU_TIMER(x) createTimer(x)
#define START_GPU_TIMER(x) startTimer(x)
#define STOP_GPU_TIMER(x) stopTimer(x)
#define GET_GPU_TIMER_ELAPSED_TIME(x, y) getTimerElapsedTime(x, y)
#define DESTROY_GPU_TIMER(x) destroyTimer(x)
#define CREATE_AND_INITIALIZE_GENERATOR(name, seed, size, buffer) \
	hiprandGenerator_t generator_##name; \
	SAFE_MALLOC_ON_DEVICE(buffer, unsigned int, size); \
	curandCheckedCall(hiprandCreateGenerator(&generator_##name, HIPRAND_RNG_PSEUDO_DEFAULT)); \
	curandCheckedCall(hiprandSetPseudoRandomGeneratorSeed(generator_##name, seed)); \
	curandCheckedCall(hiprandGeneratePoisson(generator_##name, buffer, size, 4.0))
#define DESTROY_GENERATOR(name) hiprandDestroyGenerator(generator_##name)
#else
#define SAFE_MALLOC_ON_DEVICE(__variable, __type, __amount) \
	__variable = 0; \
	__variable = (__type*)malloc(sizeof(__type) * __amount); \
	if (__variable == 0) \
	{ \
		throw std::exception(#__variable": insufficient memory"); \
	}
#define SAFE_MALLOC_PITCH_ON_DEVICE(__variable, __type, __pitch, __width, __height) \
	__variable = 0; \
	__pitch = sizeof(__type) * __width; \
	__variable = (__type*)malloc(__pitch * __height); \
	if (__variable == 0) \
	{ \
		throw std::exception(#__variable": insufficient memory"); \
	}
#define SAFE_FREE_ON_DEVICE(__variable) free(__variable)
#define MEMCPY_TO_SYMBOL(__destination, __source, __size)
#define MEMCPY_HOST_TO_DEVICE(__destination, __source, __size) memcpy(__destination, __source, __size)
#define MEMCPY2D_HOST_TO_DEVICE(__destination, __source, __hostPitch, __devicePitch, __width, __height) memcpy(__destination, __source, __hostPitch * __height)
#define MEMCPY_DEVICE_TO_DEVICE(__destination, __source, __size) memcpy(__destination, __source, __size)
#define MEMCPY_DEVICE_TO_HOST(__destination, __source, __size) memcpy(__destination, __source, __size)
#define MEMSET_ON_DEVICE(__variable, __value, __size) memset(__variable, __value, __size)
#define BIND_AS_TEXTURE2D(__deviceVariable, __texture, __pitch, __width, __height) \
	__texture.width = __width; \
	__texture.height = __height; \
	__texture.data = __deviceVariable
#define UNBIND_TEXTURE2D(__texture)
#define INVOKE_GLOBAL_CODE(__function, __numBlocks, __numThreads) __function()
#define INVOKE_GLOBAL_CODE1(__function, __numBlocks, __numThreads, __arg1) __function(__arg1)
#define INVOKE_GLOBAL_CODE2(__function, __numBlocks, __numThreads, __arg1, __arg2) __function(__arg1, __arg2)
#define INVOKE_GLOBAL_CODE3(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3) __function(__arg1, __arg2, __arg3)
#define INVOKE_GLOBAL_CODE4(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4) __function(__arg1, __arg2, __arg3, __arg4)
#define INVOKE_GLOBAL_CODE5(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5) __function(__arg1, __arg2, __arg3, __arg4, __arg5)
#define INVOKE_GLOBAL_CODE6(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6) __function(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6)
#define INVOKE_GLOBAL_CODE7(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7) __function(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7)
#define INVOKE_GLOBAL_CODE8(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8) __function(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8)
#define INVOKE_GLOBAL_CODE9(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9) __function(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9)
#define INVOKE_GLOBAL_CODE10(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10) __function(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10)
#define INVOKE_GLOBAL_CODE11(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10, __arg11) __function(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10, __arg11)
#define INVOKE_GLOBAL_CODE12(__function, __numBlocks, __numThreads, __arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10, __arg11, __arg12) __function(__arg1, __arg2, __arg3, __arg4, __arg5, __arg6, __arg7, __arg8, __arg9, __arg10, __arg11, __arg12)
#define CREATE_GPU_TIMER(x) CREATE_CPU_TIMER(x)
#define START_GPU_TIMER(x) START_CPU_TIMER(x)
#define STOP_GPU_TIMER(x) STOP_CPU_TIMER(x)
#define GET_GPU_TIMER_ELAPSED_TIME(x, y) GET_CPU_TIMER_ELAPSED_TIME(x, y)
#define DESTROY_GPU_TIMER(x) DESTROY_CPU_TIMER(x)
#define CREATE_AND_INITIALIZE_GENERATOR(name, seed, size, buffer) \
	hiprandGenerator_t generator_##name; \
	SAFE_MALLOC_ON_DEVICE(buffer, unsigned int, size); \
	curandCheckedCall(hiprandCreateGeneratorHost(&generator_##name, HIPRAND_RNG_PSEUDO_DEFAULT)); \
	curandCheckedCall(hiprandSetPseudoRandomGeneratorSeed(generator_##name, seed)); \
	curandCheckedCall(hiprandGeneratePoisson(generator_##name, buffer, size, 10.0))
#define DESTROY_GENERATOR(name) hiprandDestroyGenerator(generator_##name)
#endif

//////////////////////////////////////////////////////////////////////////
//	LOCAL DEVICE VARIABLES
//////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE Graph* dGraph;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE ImageMap* dPopulationDensityMap;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE ImageMap* dWaterBodiesMap;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE ImageMap* dBlockadesMap;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE ImageMap* dNaturalPatternMap;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE ImageMap* dRadialPatternMap;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE ImageMap* dRasterPatternMap;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE Context* dContext;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE WorkQueue* dWorkQueues1;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE WorkQueue* dWorkQueues2;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE Vertex* dVertices;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE Edge* dEdges;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE QuadTree* dQuadtree;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE Quadrant* dQuadrants;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE QuadrantEdges* dQuadrantsEdges;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE unsigned char* dPopulationDensityMapData;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE unsigned char* dWaterBodiesMapData;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE unsigned char* dBlockadesMapData;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE unsigned char* dNaturalPatternMapData;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE unsigned char* dRadialPatternMapData;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE unsigned char* dRasterPatternMapData;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE Primitive* dPrimitives;
//////////////////////////////////////////////////////////////////////////
DEVICE_VARIABLE unsigned int* dPseudoRandomNumbersBuffer;

//////////////////////////////////////////////////////////////////////////
#define allocateAndInitializeImageMap(__name1, __name2) \
	if (__name1##Map.data != 0) \
	{ \
		unsigned int hostPitch = sizeof(unsigned char) * __name1##Map.width; \
		unsigned int devicePitch; \
		SAFE_MALLOC_PITCH_ON_DEVICE(d##__name2##MapData, unsigned char, devicePitch, __name1##Map.width, __name1##Map.height); \
		BIND_AS_TEXTURE2D(d##__name2##MapData, g_d##__name2##Texture, devicePitch, __name1##Map.width, __name1##Map.height); \
		MEMCPY2D_HOST_TO_DEVICE(d##__name2##MapData, __name1##Map.data, hostPitch, devicePitch, __name1##Map.width, __name1##Map.height); \
		SAFE_MALLOC_ON_DEVICE(d##__name2##Map, ImageMap, 1); \
	}

#define deallocateImageMap(__name1, __name2) \
	if (__name1##Map.data != 0) \
	{ \
		UNBIND_TEXTURE2D(g_d##__name2##Texture); \
		SAFE_FREE_ON_DEVICE(d##__name2##MapData); \
	}

//////////////////////////////////////////////////////////////////////////
GLOBAL_CODE void initializeContext(Context* context,
								   Graph* graph,
								   ImageMap* populationDensityMap,
								   ImageMap* waterBodiesMap,
								   ImageMap* blockadesMap,
								   ImageMap* naturalPatternMap,
								   ImageMap* radialPatternMap,
								   ImageMap* rasterPatternMap,
								   Primitive* primitives,
								   unsigned int* pseudoRandomNumbersBuffer)
{
	context->graph = graph;
	context->populationDensityMap = populationDensityMap;
	context->waterBodiesMap = waterBodiesMap;
	context->blockadesMap = blockadesMap;
	context->naturalPatternMap = naturalPatternMap;
	context->radialPatternMap = radialPatternMap;
	context->rasterPatternMap = rasterPatternMap;
	context->primitives = primitives;
	context->pseudoRandomNumbersBuffer = pseudoRandomNumbersBuffer;
}

//////////////////////////////////////////////////////////////////////////
void RoadNetworkGraphGenerator::notifyObservers(Graph* graph, unsigned int numPrimitives, Primitive* primitives)
{
	for (unsigned int i = 0; i < observers.size(); i++)
	{
		observers[i]->update(graph, numPrimitives, primitives);
	}
}

//////////////////////////////////////////////////////////////////////////
void RoadNetworkGraphGenerator::copyGraphToDevice(Graph* graph)
{
	MEMCPY_HOST_TO_DEVICE(dQuadrants, graph->quadtree->quadrants, sizeof(Quadrant) * graph->quadtree->totalNumQuadrants);
	MEMCPY_HOST_TO_DEVICE(dQuadrantsEdges, graph->quadtree->quadrantsEdges, sizeof(QuadrantEdges) * graph->quadtree->numLeafQuadrants);
#ifdef COLLECT_STATISTICS
	INVOKE_GLOBAL_CODE9(updateNonPointerFields, 1, 1, dQuadtree, (int)graph->quadtree->numQuadrantEdges, graph->quadtree->worldBounds, graph->quadtree->maxDepth, graph->quadtree->totalNumQuadrants, graph->quadtree->numLeafQuadrants, (unsigned long)graph->quadtree->numCollisionChecks, (unsigned int)graph->quadtree->maxEdgesPerQuadrantInUse, (unsigned int)graph->quadtree->maxResultsPerQueryInUse);
#else
	INVOKE_GLOBAL_CODE6(updateNonPointerFields, 1, 1, dQuadtree, (int)graph->quadtree->numQuadrantEdges, graph->quadtree->worldBounds, graph->quadtree->maxDepth, graph->quadtree->totalNumQuadrants, graph->quadtree->numLeafQuadrants);
#endif
	MEMCPY_HOST_TO_DEVICE(dVertices, graph->vertices, sizeof(Vertex) * graph->numVertices);
	MEMCPY_HOST_TO_DEVICE(dEdges, graph->edges, sizeof(Edge) * graph->numEdges);
#ifdef COLLECT_STATISTICS
	INVOKE_GLOBAL_CODE4(updateNonPointerFields, 1, 1, dGraph, (int)graph->numVertices, (int)graph->numEdges, (unsigned long)graph->numCollisionChecks);
#else
	INVOKE_GLOBAL_CODE3(updateNonPointerFields, 1, 1, dGraph, (int)graph->numVertices, (int)graph->numEdges);
#endif
}

//////////////////////////////////////////////////////////////////////////
void RoadNetworkGraphGenerator::copyGraphToHost(Graph* graph)
{
	MEMCPY_DEVICE_TO_HOST(graph->quadtree->quadrants, dQuadrants, sizeof(Quadrant) * configuration.totalNumQuadrants);
	MEMCPY_DEVICE_TO_HOST(graph->quadtree->quadrantsEdges, dQuadrantsEdges, sizeof(QuadrantEdges) * configuration.numLeafQuadrants);

	Quadrant* quadrants = graph->quadtree->quadrants;
	QuadrantEdges* quadrantsEdges = graph->quadtree->quadrantsEdges;

	MEMCPY_DEVICE_TO_HOST(graph->quadtree, dQuadtree, sizeof(QuadTree));

	graph->quadtree->quadrants = quadrants;
	graph->quadtree->quadrantsEdges = quadrantsEdges;

	MEMCPY_DEVICE_TO_HOST(graph->vertices, dVertices, sizeof(Vertex) * configuration.maxVertices);
	MEMCPY_DEVICE_TO_HOST(graph->edges, dEdges, sizeof(Edge) * configuration.maxEdges);

	QuadTree* quadtree = graph->quadtree;
	Vertex* vertices = graph->vertices;
	Edge* edges = graph->edges;

	MEMCPY_DEVICE_TO_HOST(graph, dGraph, sizeof(Graph));

	graph->quadtree = quadtree;
	graph->vertices = vertices;
	graph->edges = edges;
}

//////////////////////////////////////////////////////////////////////////
void RoadNetworkGraphGenerator::execute()
{
	CREATE_AND_INITIALIZE_GENERATOR(PseudoRandomNumbers, configuration.seed, configuration.worldWidth * configuration.worldHeight, dPseudoRandomNumbersBuffer);

	CREATE_GPU_TIMER(PrimaryRoadNetworkExpansion);
	CREATE_GPU_TIMER(SecondaryRoadNetworkExpansion);
	CREATE_GPU_TIMER(GraphMemoryCopy_GpuToCpu);
	CREATE_CPU_TIMER(GraphMemoryCopy_CpuToGpu);
	CREATE_GPU_TIMER(CollisionsComputation);
	CREATE_CPU_TIMER(PrimitivesExtraction);

	allocateAndInitializeImageMap(populationDensity, PopulationDensity);
	allocateAndInitializeImageMap(waterBodies, WaterBodies);
	allocateAndInitializeImageMap(blockades, Blockades);
	allocateAndInitializeImageMap(naturalPattern, NaturalPattern);
	allocateAndInitializeImageMap(radialPattern, RadialPattern);
	allocateAndInitializeImageMap(rasterPattern, RasterPattern);

	QuadTree* quadtree;
	Quadrant* quadrants;
	QuadrantEdges* quadrantsEdges;

	SAFE_MALLOC_ON_HOST(quadtree, QuadTree, 1);
	SAFE_MALLOC_ON_HOST(quadrants, Quadrant, configuration.totalNumQuadrants);
	SAFE_MALLOC_ON_HOST(quadrantsEdges, QuadrantEdges, configuration.numLeafQuadrants);

	memset(quadrants, 0, sizeof(Quadrant) * configuration.totalNumQuadrants);
	memset(quadrantsEdges, 0, sizeof(QuadrantEdges) * configuration.numLeafQuadrants);

	SAFE_MALLOC_ON_DEVICE(dQuadtree, QuadTree, 1);
	SAFE_MALLOC_ON_DEVICE(dQuadrants, Quadrant, configuration.totalNumQuadrants);
	SAFE_MALLOC_ON_DEVICE(dQuadrantsEdges, QuadrantEdges, configuration.numLeafQuadrants);

	MEMSET_ON_DEVICE(dQuadrants, 0, sizeof(Quadrant) * configuration.totalNumQuadrants);
	MEMSET_ON_DEVICE(dQuadrantsEdges, 0, sizeof(QuadrantEdges) * configuration.numLeafQuadrants);

	Box2D worldBounds(0.0f, 0.0f, (float)configuration.worldWidth, (float)configuration.worldHeight);

	initializeQuadtreeOnHost(quadtree, worldBounds, configuration.quadtreeDepth, configuration.totalNumQuadrants, configuration.numLeafQuadrants, quadrants, quadrantsEdges);
	INVOKE_GLOBAL_CODE7(initializeQuadtreeOnDevice, 1, 1, dQuadtree, worldBounds, configuration.quadtreeDepth, configuration.totalNumQuadrants, configuration.numLeafQuadrants, dQuadrants, dQuadrantsEdges);

	Graph* graph;
	Vertex* vertices;
	Edge* edges;

	SAFE_MALLOC_ON_HOST(graph, Graph, 1);
	SAFE_MALLOC_ON_HOST(vertices, Vertex, configuration.maxVertices);
	SAFE_MALLOC_ON_HOST(edges, Edge, configuration.maxEdges);

	memset(vertices, 0, sizeof(Vertex) * configuration.maxVertices);
	memset(edges, 0, sizeof(Edge) * configuration.maxEdges);

	SAFE_MALLOC_ON_DEVICE(dGraph, Graph, 1);
	SAFE_MALLOC_ON_DEVICE(dVertices, Vertex, configuration.maxVertices);
	SAFE_MALLOC_ON_DEVICE(dEdges, Edge, configuration.maxEdges);

	MEMSET_ON_DEVICE(dVertices, 0, sizeof(Vertex) * configuration.maxVertices);
	MEMSET_ON_DEVICE(dEdges, 0, sizeof(Edge) * configuration.maxEdges);
	
	initializeGraphOnHost(graph, configuration.snapRadius, configuration.maxVertices, configuration.maxEdges, vertices, edges, quadtree);
	INVOKE_GLOBAL_CODE7(initializeGraphOnDevice, 1, 1, dGraph, configuration.snapRadius, configuration.maxVertices, configuration.maxEdges, dVertices, dEdges, dQuadtree);

	SAFE_MALLOC_ON_DEVICE(dWorkQueues1, WorkQueue, NUM_PROCEDURES);
	SAFE_MALLOC_ON_DEVICE(dWorkQueues2, WorkQueue, NUM_PROCEDURES);

	WorkQueue* workQueues1;
	WorkQueue* workQueues2;

	SAFE_MALLOC_ON_HOST(workQueues1, WorkQueue, NUM_PROCEDURES);
	SAFE_MALLOC_ON_HOST(workQueues2, WorkQueue, NUM_PROCEDURES);

	memset(workQueues1, 0, sizeof(WorkQueue) * NUM_PROCEDURES);
	memset(workQueues2, 0, sizeof(WorkQueue) * NUM_PROCEDURES);

	// set highway spawn points
	for (unsigned int i = 0; i < configuration.numSpawnPoints; i++)
	{
		vml_vec2 spawnPoint = configuration.getSpawnPoint(i);
		int source = createVertex(graph, spawnPoint);
		workQueues1[EVALUATE_HIGHWAY].unsafePush(Highway(RoadAttributes(source, configuration.highwayLength, 0), UNASSIGNED));
		workQueues1[EVALUATE_HIGHWAY].unsafePush(Highway(RoadAttributes(source, configuration.highwayLength, -HALF_PI), UNASSIGNED));
		workQueues1[EVALUATE_HIGHWAY].unsafePush(Highway(RoadAttributes(source, configuration.highwayLength, HALF_PI), UNASSIGNED));
		workQueues1[EVALUATE_HIGHWAY].unsafePush(Highway(RoadAttributes(source, configuration.highwayLength, PI), UNASSIGNED));
	}

	SAFE_MALLOC_ON_DEVICE(dPrimitives, Primitive, configuration.maxPrimitives);

	START_CPU_TIMER(GraphMemoryCopy_CpuToGpu);

	copyGraphToDevice(graph);

	MEMCPY_HOST_TO_DEVICE(dWorkQueues1, workQueues1, sizeof(WorkQueue) * NUM_PROCEDURES);
	MEMCPY_HOST_TO_DEVICE(dWorkQueues2, workQueues2, sizeof(WorkQueue) * NUM_PROCEDURES);
	
#ifdef USE_CUDA
	MEMCPY_TO_SYMBOL(g_dConfiguration, &configuration, sizeof(Configuration)); 
#else
	g_dConfiguration = configuration;
#endif

	STOP_CPU_TIMER(GraphMemoryCopy_CpuToGpu);

	SAFE_MALLOC_ON_DEVICE(dContext, Context, 1);
	INVOKE_GLOBAL_CODE10(initializeContext, 1, 1, 
		dContext,
		dGraph, 
		dPopulationDensityMap,
		dWaterBodiesMap,
		dBlockadesMap,
		dNaturalPatternMap,
		dRadialPatternMap,
		dRasterPatternMap,
		dPrimitives,
		dPseudoRandomNumbersBuffer);

	START_GPU_TIMER(PrimaryRoadNetworkExpansion);

	// expand primary road network
	expand(configuration.maxHighwayDerivation, 0, 3);

	STOP_GPU_TIMER(PrimaryRoadNetworkExpansion);

	Log::logger("default") << "primary road network expansion: " << elapsedTime_PrimaryRoadNetworkExpansion << " (ms)" << Logger::endl;

	START_GPU_TIMER(CollisionsComputation);

	computeCollisions();

	STOP_GPU_TIMER(CollisionsComputation);

	Log::logger("default") << "collisions computation: " << elapsedTime_CollisionsComputation << " (ms)" << Logger::endl;

	START_GPU_TIMER(GraphMemoryCopy_GpuToCpu);

	copyGraphToHost(graph);

	STOP_GPU_TIMER(GraphMemoryCopy_GpuToCpu);

	BaseGraph* graphCopy;
	Vertex* verticesCopy;
	Edge* edgesCopy;

	SAFE_MALLOC_ON_HOST(graphCopy, BaseGraph, 1);
	SAFE_MALLOC_ON_HOST(verticesCopy, Vertex, graph->numVertices);
	SAFE_MALLOC_ON_HOST(edgesCopy, Edge, graph->numEdges);

	graphCopy->vertices = verticesCopy;
	graphCopy->edges = edgesCopy;

	memcpy(graphCopy->vertices, graph->vertices, sizeof(Vertex) * graph->numVertices);
	memcpy(graphCopy->edges, graph->edges, sizeof(Edge) * graph->numEdges);
	
	graphCopy->numVertices = graph->numVertices;
	graphCopy->numEdges = graph->numEdges;

	Primitive* primitives;
	
	SAFE_MALLOC_ON_HOST(primitives, Primitive, configuration.maxPrimitives);

	memset(primitives, 0, sizeof(Primitive) * configuration.maxPrimitives);

	// extract the city cells

	START_CPU_TIMER(PrimitivesExtraction);

	unsigned int numPrimitives = extractPrimitives(graphCopy, primitives, configuration.maxPrimitives);

	STOP_CPU_TIMER(PrimitivesExtraction);

	Log::logger("default") << "primitives extraction: " << elapsedTime_PrimitivesExtraction << " (ms)" << Logger::endl;

	free(graphCopy);
	free(verticesCopy);
	free(edgesCopy);

	memset(workQueues1, 0, sizeof(WorkQueue) * NUM_PROCEDURES);
	memset(workQueues2, 0, sizeof(WorkQueue) * NUM_PROCEDURES);

	for (unsigned int i = 0; i < numPrimitives; i++)
	{
		Primitive& primitive = primitives[i];

		// remove filaments contained by minimal cycles
		if (primitive.type == FILAMENT)
		{
			for (unsigned int j = 0; j < numPrimitives; j++)
			{
				Primitive& otherPrimitive = primitives[j];

				if (otherPrimitive.type != MINIMAL_CYCLE)
				{
					continue;
				}

				for (unsigned int k = 0; k < primitive.numVertices; k++)
				{
					if (MathExtras::inside(otherPrimitive.vertices, otherPrimitive.numVertices, primitive.vertices[k]))
					{
						primitive.removed = true;
						break;
					}
				}

				if (primitive.removed)
				{
					break;
				}
			}
		}
		// set street spawn points
		else if (primitive.type == MINIMAL_CYCLE)
		{
			for (unsigned int j = 0; j < primitive.numEdges; j++)
			{
				Edge& edge = graph->edges[primitive.edges[j]];

				// FIXME: checking invariants
				if (edge.numPrimitives >= 2)
				{
					THROW_EXCEPTION("edge.numPrimitives >= 2");
				}

				edge.primitives[edge.numPrimitives++] = i;
			}

			vml_vec2 centroid;
			float area;
			MathExtras::getPolygonInfo(primitive.vertices, primitive.numVertices, area, centroid);
			if (area < configuration.minBlockArea)
			{
				continue;
			}

			if (!MathExtras::inside(primitive.vertices, primitive.numVertices, centroid))
			{
				continue;
			}

			float angle;
			// FIXME: enforce primitive convex hull
			ConvexHull convexHull(primitive.vertices, primitive.numVertices);
			OBB2D obb(convexHull.hullPoints, convexHull.numHullPoints);
			angle = vml_angle(obb.axis[1], vml_vec2(0.0f, 1.0f));

			int source = createVertex(graph, centroid);
			workQueues1[EVALUATE_STREET].unsafePush(Street(RoadAttributes(source, configuration.streetLength, angle), StreetRuleAttributes(0, i), UNASSIGNED));
			workQueues1[EVALUATE_STREET].unsafePush(Street(RoadAttributes(source, configuration.streetLength, -HALF_PI + angle), StreetRuleAttributes(0, i), UNASSIGNED));
			workQueues1[EVALUATE_STREET].unsafePush(Street(RoadAttributes(source, configuration.streetLength, HALF_PI + angle), StreetRuleAttributes(0, i), UNASSIGNED));
			workQueues1[EVALUATE_STREET].unsafePush(Street(RoadAttributes(source, configuration.streetLength, PI + angle), StreetRuleAttributes(0, i), UNASSIGNED));
		} 
		// remove isolated vertices
		else
		{
			primitive.removed = true;
		}
	}

	START_CPU_TIMER(GraphMemoryCopy_CpuToGpu);
	
	MEMCPY_HOST_TO_DEVICE(dPrimitives, primitives, sizeof(Primitive) * numPrimitives);

	copyGraphToDevice(graph);

	MEMCPY_HOST_TO_DEVICE(dWorkQueues1, workQueues1, sizeof(WorkQueue) * NUM_PROCEDURES);
	MEMCPY_HOST_TO_DEVICE(dWorkQueues2, workQueues2, sizeof(WorkQueue) * NUM_PROCEDURES);

	STOP_CPU_TIMER(GraphMemoryCopy_CpuToGpu);

	START_GPU_TIMER(SecondaryRoadNetworkExpansion);

	// expand secondary road network
	expand(configuration.maxStreetDerivation, 3, 2);

	STOP_GPU_TIMER(SecondaryRoadNetworkExpansion);

	Log::logger("default") << "secondary road network expansion: " << elapsedTime_SecondaryRoadNetworkExpansion << " (ms)" << Logger::endl;

	START_GPU_TIMER(GraphMemoryCopy_GpuToCpu);

	copyGraphToHost(graph);
	MEMCPY_DEVICE_TO_HOST(primitives, dPrimitives, sizeof(Primitive) * configuration.maxPrimitives);

	STOP_GPU_TIMER(GraphMemoryCopy_GpuToCpu);

	Log::logger("default") << "graph memory copy (gpu -> cpu): " << elapsedTime_GraphMemoryCopy_GpuToCpu << " (ms)" << Logger::endl;
	Log::logger("default") << "graph memory copy (cpu -> gpu): " << elapsedTime_GraphMemoryCopy_CpuToGpu << " (ms)" << Logger::endl;

#ifdef COLLECT_STATISTICS
	maxPrimitiveSize = 0;
	for (unsigned int i = 0; i < numPrimitives; i++)
	{
		maxPrimitiveSize = MathExtras::max(maxPrimitiveSize, primitives[i].numEdges);
	}

	unsigned int numPrimaryRoadnetworkEdges = 0;
	unsigned int numSecondaryRoadnetworkEdges = 0;
	for (unsigned int i = 0; i < graph->numEdges; i++)
	{
		Edge& edge = graph->edges[i];
		if (edge.attr1 == 0)
		{
			numSecondaryRoadnetworkEdges++;
		}
		else if (edge.attr1 == 1)
		{
			numPrimaryRoadnetworkEdges++;
		}
		else
		{
			// FIXME: checking invariants
			THROW_EXCEPTION1("unknown edge attr1 value (%d)", edge.attr1);
		}
	}

	unsigned long numCollisionChecks = graph->numCollisionChecks + quadtree->numCollisionChecks;
	unsigned int memoryInUse = getMemoryInUse(graph) + getMemoryInUse(quadtree);

	Log::logger("default") << "vertices (alloc./in use): " << graph->maxVertices << " / " << graph->numVertices << Logger::endl;
	Log::logger("default") << "edges (alloc./in use): " << graph->maxEdges << " / " << graph->numEdges << Logger::endl;
	Log::logger("default") << "vertex in connections (alloc./max. in use): " << MAX_VERTEX_IN_CONNECTIONS << " / " << getMaxVertexInConnectionsInUse(graph) << Logger::endl;
	Log::logger("default") << "vertex out connections (alloc./max. in use): " << MAX_VERTEX_OUT_CONNECTIONS << " / " << getMaxVertexOutConnectionsInUse(graph) << Logger::endl;
	Log::logger("default") << "avg. vertex in connections (in use): " << getAverageVertexInConnectionsInUse(graph) << Logger::endl;
	Log::logger("default") << "avg. vertex out connections (in use): " << getAverageVertexOutConnectionsInUse(graph) << Logger::endl;
	Log::logger("default") << "num. primitives (alloc./in use): " << configuration.maxPrimitives << " / " << numPrimitives << Logger::endl;
	Log::logger("default") << "num. primitive edges (alloc./max. in use): " << MAX_EDGES_PER_PRIMITIVE << " / " << maxPrimitiveSize << Logger::endl;
	Log::logger("default") << "edges per quadrant (alloc./max. in use): " << MAX_EDGES_PER_QUADRANT << " / " << quadtree->maxEdgesPerQuadrantInUse << Logger::endl;
	Log::logger("default") << "memory (alloc./in use): " << toMegabytes(getAllocatedMemory(graph) + getAllocatedMemory(quadtree)) << " mb / " << toMegabytes(memoryInUse) << " mb" << Logger::endl;
	Log::logger("default") << "num. collision checks: " << numCollisionChecks << Logger::endl;

	if (g_dumpStatistics)
	{
		if (Log::logger("statistics").firstUse())
		{
			// header
			Log::logger("statistics") << "timestamp" 
				<< "config_name" 
				<< "expansion_kernel_blocks" 
				<< "expansion_kernel_threads" 
				<< "collision_detection_kernel_blocks" 
				<< "collision_detection_kernel_threads" 
				<< "max_highway_derivations"
				<< "max_street_derivations"
				<< "quadtree_depth"
				<< "primary_roadnetwork_expansion_time" 
				<< "collisions_computation_time" 
				<< "primitives_extraction_time" 
				<< "secondary_roadnetwork_expansion_time" 
				<< "memory_copy_gpu_cpu_time" 
				<< "memory_copy_cpu_gpu_time" 
				<< "num_vertices" 
				<< "num_primary_roadnetwork_edges" 
				<< "num_secondary_roadnetwork_edges" 
				<< "num_collisions" 
				<< "memory_in_use" 
				<< Logger::endl;
		}
		Log::logger("statistics") << Timer::getTimestamp() 
			<< configuration.name 
			<< configuration.numExpansionKernelBlocks 
			<< configuration.numExpansionKernelThreads 
			<< configuration.numLeafQuadrants 
			<< configuration.numCollisionDetectionKernelThreads
			<< configuration.maxHighwayDerivation
			<< configuration.maxStreetDerivation
			<< configuration.quadtreeDepth
			<< elapsedTime_PrimaryRoadNetworkExpansion 
			<< elapsedTime_CollisionsComputation
			<< elapsedTime_PrimitivesExtraction
			<< elapsedTime_SecondaryRoadNetworkExpansion 
			<< elapsedTime_GraphMemoryCopy_GpuToCpu 
			<< elapsedTime_GraphMemoryCopy_CpuToGpu 
			<< graph->numVertices 
			<< numPrimaryRoadnetworkEdges 
			<< numSecondaryRoadnetworkEdges
			<< numCollisionChecks 
			<< memoryInUse 
			<< Logger::endl;
	}
#endif

	notifyObservers(graph, numPrimitives, primitives);

	SAFE_FREE_ON_DEVICE(dContext);
	SAFE_FREE_ON_DEVICE(dWorkQueues1);
	SAFE_FREE_ON_DEVICE(dWorkQueues2);
	SAFE_FREE_ON_DEVICE(dPopulationDensityMap);
	SAFE_FREE_ON_DEVICE(dWaterBodiesMap);
	SAFE_FREE_ON_DEVICE(dBlockadesMap);
	SAFE_FREE_ON_DEVICE(dNaturalPatternMap);
	SAFE_FREE_ON_DEVICE(dRadialPatternMap);
	SAFE_FREE_ON_DEVICE(dRasterPatternMap);
	deallocateImageMap(populationDensity, PopulationDensity);
	deallocateImageMap(waterBodies, WaterBodies);
	deallocateImageMap(blockades, Blockades);
	deallocateImageMap(naturalPattern, NaturalPattern);
	deallocateImageMap(radialPattern, RadialPattern);
	deallocateImageMap(rasterPattern, RasterPattern);
	SAFE_FREE_ON_DEVICE(dPrimitives);

	SAFE_FREE_ON_DEVICE(dQuadrants);
	SAFE_FREE_ON_DEVICE(dQuadrantsEdges);
	SAFE_FREE_ON_DEVICE(dQuadtree);
	SAFE_FREE_ON_DEVICE(dVertices);
	SAFE_FREE_ON_DEVICE(dEdges);
	SAFE_FREE_ON_DEVICE(dGraph);

	free(primitives);
	free(workQueues1);
	free(workQueues2);
	free(quadrants);
	free(quadrantsEdges);
	free(quadtree);
	free(graph);
	free(vertices);
	free(edges);

	DESTROY_GPU_TIMER(PrimaryRoadNetworkExpansion);
	DESTROY_GPU_TIMER(SecondaryRoadNetworkExpansion);
	DESTROY_GPU_TIMER(GraphMemoryCopy_GpuToCpu);
	DESTROY_CPU_TIMER(PrimitivesExtraction);

	DESTROY_GENERATOR(PseudoRandomNumbers);
	SAFE_FREE_ON_DEVICE(dPseudoRandomNumbersBuffer);
}

#ifdef USE_CUDA
//////////////////////////////////////////////////////////////////////////
void RoadNetworkGraphGenerator::expand(unsigned int numDerivations, unsigned int startingQueue, unsigned int numQueues)
{
	WorkQueue* frontQueues = dWorkQueues1;
	WorkQueue* backQueues = dWorkQueues2;
	for (unsigned int i = 0; i < numDerivations; i++)
	{
		expansionKernel<<<configuration.numExpansionKernelBlocks, configuration.numExpansionKernelThreads>>>(frontQueues, backQueues, startingQueue, numQueues, dContext);
		cudaCheckError();
		WorkQueue* tmp = frontQueues;
		frontQueues = backQueues;
		backQueues = tmp;
	}
}

//////////////////////////////////////////////////////////////////////////
void RoadNetworkGraphGenerator::computeCollisions()
{
	collisionDetectionKernel<<<configuration.numLeafQuadrants, configuration.numCollisionDetectionKernelThreads>>>(dGraph);
	cudaCheckError();
}

#else
//////////////////////////////////////////////////////////////////////////
void RoadNetworkGraphGenerator::expand(unsigned int numDerivations, unsigned int startingQueue, unsigned int numQueues)
{
	expansionKernel(numDerivations, dWorkQueues1, dWorkQueues2, startingQueue, numQueues, dContext);
}

//////////////////////////////////////////////////////////////////////////
void RoadNetworkGraphGenerator::computeCollisions()
{
	collisionDetectionKernel(dGraph);
}

#endif